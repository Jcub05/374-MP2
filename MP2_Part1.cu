#include "hip/hip_runtime.h"
﻿// Jacob Badali 20290739
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <stdlib.h>


#define WIDTH (1500) //CHANGE THIS!!!

#define BLOCK_WIDTH 2 //CHANGE THIS!!!



//Multiplication kernel function
__global__ void mulKernel(float* M, float* N, float* P, int size) {
    int rows = blockIdx.y * blockDim.y + threadIdx.y;
    int cols = blockIdx.x * blockDim.x + threadIdx.x;


    if (rows < size && cols < size) {
        float temp_sum = 0.0;
        for (int i = 0; i < size; i++) {
            temp_sum += M[rows * size + i] * N[i * size + cols];
        }
        P[rows * size + cols] = temp_sum;
    }
}


int main()
{

    float* d_M = 0;
    float* d_N = 0;
    float* d_P = 0;

    float* h_M;
    float* h_N;
    float* h_P;
    float* h_Pcheck;


    int size = WIDTH * WIDTH * sizeof(float);

    hipHostMalloc((void**)&h_M, size);
    hipHostMalloc((void**)&h_N, size);
    hipHostMalloc((void**)&h_P, size);
    hipHostMalloc((void**)&h_Pcheck, size);

    int NumBlocks = WIDTH / BLOCK_WIDTH;
    if (WIDTH % BLOCK_WIDTH) NumBlocks++;

    dim3 dimGrid(NumBlocks, NumBlocks);
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float elapsedTime_DevToHost;
    float elapsedTime_HostToDev;
    float elapsedTime_MatrixMulHost;
    float elapsedTime_MatrixMulDev;

    //Allocate appropriate memory size for each array
    hipMalloc((void**)&d_M, size);
    hipMalloc((void**)&d_N, size);
    hipMalloc((void**)&d_P, size);

    //fill host matrices
    for (int k = 0; k < WIDTH; k++) {
        for (int j = 0; j < WIDTH; j++) {
            h_M[k * WIDTH + j] = ((float)rand() / RAND_MAX) * 100.0f; // fill with rand values from 0-100
            h_N[k * WIDTH + j] = ((float)rand() / RAND_MAX) * 100.0f;
            h_P[k * WIDTH + j] = 0.0;
            h_Pcheck[k * WIDTH + j] = 0.0;
        }
    }

    

    //Cpy to dev, timer
    hipEventRecord(start, 0);
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime_HostToDev, start, stop);
    printf("Transfer Host to Device, size[%d]: %f ms |", WIDTH, elapsedTime_HostToDev);

    hipEventRecord(start, 0);
    hipMemcpy(h_M, d_M, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_N, d_N, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTidme_DevToHost, start, stop);
    printf("| Transfer Device to Host, size[%d]: %f ms\n", WIDTH, elapsedTime_DevToHost);

    printf("\n");

    //Host Matrix Multiplication
    hipEventRecord(start, 0);


    //for (int i = 0; i < WIDTH; i++) {
    //    for (int j = 0; j < WIDTH; j++) {
    //        for (int k = 0; k < WIDTH; k++) {
    //            h_Pcheck[i * WIDTH + j] += h_M[i * WIDTH + k] * h_N[k * WIDTH + j];
    //        }
    //    }
    //}

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime_MatrixMulHost, start, stop);
    printf("Host Matrix Mul Time, size[%d]: %f ms\n", WIDTH, elapsedTime_MatrixMulHost);

    //Device Matrix Multiplication
    hipEventRecord(start, 0);
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventRecord(start, 0);

    mulKernel << <dimGrid, dimBlock, 0, 0>> > (d_M, d_N, d_P, WIDTH);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime_MatrixMulDev, start, stop);
    printf("Device Matrix Mul Time, size[%d]: %f ms\n", WIDTH, elapsedTime_MatrixMulDev);

    hipMemcpy(h_M, d_M, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_N, d_N, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 

    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);

    int check = 0;
    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < WIDTH; j++) {
            if (abs(h_P[i * WIDTH + j] - h_Pcheck[i * WIDTH + j]) > 1) {
                check = 1;
            }
        }
    }

    if (check == 0) {
        printf("TEST PASSED\n");
    }
    else {
        printf("TEST FAILED\n");
    }

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
    hipFree(h_M);
    hipFree(h_N);
    hipFree(h_P);
    hipFree(h_Pcheck);
}