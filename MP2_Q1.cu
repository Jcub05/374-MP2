﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

// Helper function to convert compute capability to the number of cores
int ConvertSMVer2Cores(int major, int minor) {
    // Refer to NVIDIA CUDA Programming Guide for the compute capability to cores conversion
    // This is a simplified version and may not cover all cases
    int cores;

    switch ((major << 4) + minor) {
    case 0x10:
        cores = 8;
        break;
    case 0x11:
    case 0x12:
        cores = 8;
        break;
    case 0x13:
        cores = 32;
        break;
    case 0x20:
        cores = 32;
        break;
    default:
        cores = 0;
        break;
    }

    return cores;
}


int main() {
    int device_id = 0; // Device ID (you can change it if you have multiple devices)
    hipSetDevice(device_id); // Set the device to use

    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, device_id);

    int num_SM = device_prop.multiProcessorCount;
    int max_threads_per_SM = device_prop.maxThreadsPerMultiProcessor;
    int warp_size = device_prop.warpSize;

    int threads_scheduled = num_SM * (max_threads_per_SM / warp_size);

    printf("Number of Streaming Multiprocessors: %d\n", num_SM);
    printf("Max Threads per Multiprocessor: %d\n", max_threads_per_SM);
    printf("Warp Size: %d\n", warp_size);
    printf("Threads Scheduled: %d\n", threads_scheduled);

    return 0;
}